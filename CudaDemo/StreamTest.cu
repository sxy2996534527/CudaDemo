#include "hip/hip_runtime.h"
#include "StreamTest.cuh"

#include <stdio.h>
#include <iostream>

__global__ void SimpleAddKernel(float* outputDevPtr, float* inputDevPtr, int data_size) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < data_size)
        outputDevPtr[i] = inputDevPtr[i] + 1;
}

void CUDART_CB MyCallback(void* data) {
    printf("Inside callback %d\n", (size_t)data);
}

StreamTest::StreamTest() {
    m_data_size = 51200;
}

StreamTest::~StreamTest() {

}

hipError_t StreamTest::CreateStream() {
    hipError_t cuda_status;

    for (int i = 0; i < 2; ++i)
        hipStreamCreate(&m_stream[i]);
    //float* hostPtr;
    //hipHostMalloc(&hostPtr, 2 * size);

    cuda_status = hipGetLastError();
    if (cuda_status != hipSuccess) {
        fprintf(stderr, "Create stream failed: %s\n", hipGetErrorString(cuda_status));
        goto Error;
    }

Error:
    return cuda_status;
}

hipError_t StreamTest::DestroyStream() {
    hipError_t cuda_status;

    for (int i = 0; i < 2; ++i)
        hipStreamDestroy(m_stream[i]);
    //float* hostPtr;
    //hipHostMalloc(&hostPtr, 2 * size);

    cuda_status = hipGetLastError();
    if (cuda_status != hipSuccess) {
        fprintf(stderr, "Destroy stream failed: %s\n", hipGetErrorString(cuda_status));
        goto Error;
    }

Error:
    return cuda_status;
}

hipError_t StreamTest::SetL2PersistingAccessAttr(int stream_id) {
    hipError_t cuda_status;

    hipDeviceProp_t prop;                                                                        // CUDA device properties variable
    cuda_status = hipGetDeviceProperties(&prop, 0);                                                 // Query GPU properties
    if (cuda_status != hipSuccess) {
        fprintf(stderr, "hipGetDeviceProperties failed!");
        goto Error;
    }
    
    size_t l2_size = min(int(prop.l2CacheSize * 0.75), prop.persistingL2CacheMaxSize);
    cuda_status = hipDeviceSetLimit(cudaLimitPersistingL2CacheSize, l2_size);                                  // set-aside 3/4 of L2 cache for persisting accesses or the max allowed
    if (cuda_status != hipSuccess) {
        fprintf(stderr, "hipDeviceSetLimit failed!");
        goto Error;
    }

    int num_bytes = 512;
    size_t window_size = min(prop.accessPolicyMaxWindowSize, num_bytes);                        // Select minimum of user defined num_bytes and max window size.

    //size_t data_size = size * sizeof(float);
    //// Allocate vectors in device memory
    //float* d_A;
    //cuda_status = hipMalloc(&d_A, data_size);
    //// Copy vectors from host memory to device memory
    //cuda_status = hipMemcpy(d_A, h_data, data_size, hipMemcpyHostToDevice);

    hipLaunchAttributeValue stream_attribute;                                                       // Stream level attributes data structure
    //stream_attribute.accessPolicyWindow.base_ptr = reinterpret_cast<void*>(d_A);               // Global Memory data pointer
    stream_attribute.accessPolicyWindow.num_bytes = window_size;                                // Number of bytes for persistence access
    stream_attribute.accessPolicyWindow.hitRatio = 0.6;                                        // Hint for cache hit ratio
    stream_attribute.accessPolicyWindow.hitProp = hipAccessPropertyPersisting;               // Persistence Property
    stream_attribute.accessPolicyWindow.missProp = hipAccessPropertyStreaming;                // Type of access property on cache miss

    cuda_status = hipStreamSetAttribute(m_stream[stream_id], hipLaunchAttributeAccessPolicyWindow, &stream_attribute);   // Set the attributes to a CUDA Stream
    if (cuda_status != hipSuccess) {
        fprintf(stderr, "hipStreamSetAttribute failed!");
        goto Error;
    }
    //for (int i = 0; i < 10; i++) {
    //    cuda_kernelA << <grid_size, block_size, 0, stream >> > (data1);                                 // This data1 is used by a kernel multiple times
    //}                                                                                           // [data1 + num_bytes) benefits from L2 persistence
    //cuda_kernelB << <grid_size, block_size, 0, stream >> > (data1);                                     // A different kernel in the same stream can also benefit
    //                                                                                            // from the persistence of data1

    //stream_attribute.accessPolicyWindow.num_bytes = 0;                                          // Setting the window size to 0 disable it
    //hipStreamSetAttribute(stream, hipLaunchAttributeAccessPolicyWindow, &stream_attribute);   // Overwrite the access policy attribute to a CUDA Stream
    //cudaCtxResetPersistingL2Cache();                                                            // Remove any persistent lines in L2 

    //cuda_kernelC << <grid_size, block_size, 0, stream >> > (data2);                                     // data2 can now benefit from full L2 in normal mode

Error:
    return cuda_status;
}

hipError_t StreamTest::AsyncMemoryCopy() {
    hipError_t cuda_status;

    size_t data_size = m_data_size * sizeof(float);
    cuda_status = hipHostMalloc(&m_host_data, 2 * data_size);

    // Assign values to the pinned memory
    for (size_t i = 0; i < 2 * m_data_size; i++) {
        m_host_data[i] = static_cast<float>(i * 1.5f);
    }


    float* inputDevPtr;
    float* outputDevPtr;
    cuda_status = hipMalloc(&inputDevPtr, 2 * data_size);
    cuda_status = hipMalloc(&outputDevPtr, 2 * data_size);

    for (int i = 0; i < 2; ++i) {
        cuda_status = hipMemcpyAsync(inputDevPtr + i * m_data_size, m_host_data + i * m_data_size,
            data_size, hipMemcpyHostToDevice, m_stream[i]);
        SimpleAddKernel << <100, 512, 0, m_stream[i] >> >
            (outputDevPtr + i * m_data_size, inputDevPtr + i * m_data_size, m_data_size);
        cuda_status = hipMemcpyAsync(m_host_data + i * m_data_size, outputDevPtr + i * m_data_size,
            data_size, hipMemcpyDeviceToHost, m_stream[i]);
    }
    //cuda_status = hipMemcpyAsync(inputDevPtr, m_host_data,
    //    data_size, hipMemcpyHostToDevice, m_stream[0]);
    //SimpleAddKernel << <100, 512, 0, m_stream[0] >> >
    //    (outputDevPtr, inputDevPtr, m_data_size);
    //cuda_status = hipMemcpyAsync(m_host_data, outputDevPtr,
    //    data_size, hipMemcpyDeviceToHost, m_stream[0]);

    cuda_status = hipGetLastError();
    if (cuda_status != hipSuccess) {
        fprintf(stderr, "Execute kernel failed: %s\n", hipGetErrorString(cuda_status));
        goto Error;
    }

Error:
    hipHostFree(m_host_data);
    hipFree(inputDevPtr);
    hipFree(outputDevPtr);
    return cuda_status;
}

hipError_t StreamTest::LaunchHostFuncInStream() {
    hipError_t cuda_status;

    size_t data_size = m_data_size * sizeof(float);

    float** hostPtr = new float* [2];
    for (int i = 0; i < 2; i++) {
        cuda_status = hipHostMalloc(&hostPtr[i], data_size);
    }

    // Assign values to the pinned memory
    for (size_t i = 0; i < m_data_size; i++) {
        hostPtr[0][i] = static_cast<float>(i * 0.5f);
        hostPtr[1][i] = static_cast<float>(i * 1.5f);
    }

    float** devPtrIn = new float*[2];
    float** devPtrOut = new float* [2];
    for (int i = 0; i < 2; i++) {
        cuda_status = hipMalloc(&devPtrIn[i], data_size);
        cuda_status = hipMalloc(&devPtrOut[i], data_size);
    }

    for (size_t i = 0; i < 2; ++i) {
        hipMemcpyAsync(devPtrIn[i], hostPtr[i], data_size, hipMemcpyHostToDevice, m_stream[i]);
        SimpleAddKernel << <100, 512, 0, m_stream[i] >> > (devPtrOut[i], devPtrIn[i], m_data_size);
        hipMemcpyAsync(hostPtr[i], devPtrOut[i], data_size, hipMemcpyDeviceToHost, m_stream[i]);
        hipLaunchHostFunc(m_stream[i], MyCallback, (void*)i);
    }


    cuda_status = hipGetLastError();
    if (cuda_status != hipSuccess) {
        fprintf(stderr, "LaunchHostFuncInStream: %s\n", hipGetErrorString(cuda_status));
        goto Error;
    }

Error:
    for (int i = 0; i < 2; i++) {
        hipHostFree(hostPtr[i]);
        hipFree(devPtrIn[i]);
        hipFree(devPtrOut[i]);
    }

    return cuda_status;
}

hipError_t StreamTest::CreateStreamWithPriority() {
    hipError_t cuda_status;
    // get the range of stream priorities for this device
    int priority_high, priority_low;
    hipDeviceGetStreamPriorityRange(&priority_low, &priority_high);
    // create streams with highest and lowest available priorities
    hipStreamCreateWithPriority(&m_stream[0], hipStreamNonBlocking, priority_high);
    hipStreamCreateWithPriority(&m_stream[1], hipStreamNonBlocking, priority_low);
    cuda_status = hipGetLastError();
    if (cuda_status != hipSuccess) {
        fprintf(stderr, "LaunchHostFuncInStream: %s\n", hipGetErrorString(cuda_status));
        goto Error;
    }
Error:
    //hipStreamDestroy(m_stream[0]);
    //hipStreamDestroy(m_stream[1]);
    return cuda_status;
}

int StreamTest::TestSimpleAsyncStream() {
    hipError_t cuda_status = CreateStream();
    if (cuda_status != hipSuccess) {
        fprintf(stderr, "CreateStream failed: %s\n", hipGetErrorString(cuda_status));
        return 1;
    }

    cuda_status = AsyncMemoryCopy();
    if (cuda_status != hipSuccess) {
        fprintf(stderr, "TestSimpleAsyncStream failed: %s\n", hipGetErrorString(cuda_status));
        return 1;
    }

    return 0;
}

int StreamTest::CountPinnedMem() {
    size_t stepSize = 100 * 1024 * 1024;  // Start with 100 MB increments
    size_t maxAllocSize = 0;
    float* pinnedMemory = nullptr;

    int flag = 0;
    while (!flag) {
        hipError_t err = hipHostMalloc((void**)&pinnedMemory, maxAllocSize + stepSize);
        if (err == hipSuccess) {
            std::cout << "Already allocated pinned memory: " << maxAllocSize / (1024.0 * 1024.0) << " MB" << std::endl;
            maxAllocSize += stepSize;
            hipHostFree(pinnedMemory);  // Free the memory after successful allocation
        }
        else {
            std::cout << "Failed to allocate additional pinned memory." << std::endl;
            flag = 1;
        }
    }

    std::cout << "Approximate maximum pinned memory: " << maxAllocSize / (1024.0 * 1024.0) << " MB" << std::endl;

    return 0;
}

int StreamTest::TestLaunchHostFuncInStream() {
    hipError_t cuda_status = CreateStream();
    if (cuda_status != hipSuccess) {
        fprintf(stderr, "CreateStream failed: %s\n", hipGetErrorString(cuda_status));
        return 1;
    }

    cuda_status = LaunchHostFuncInStream();
    if (cuda_status != hipSuccess) {
        fprintf(stderr, "TestLaunchHostFuncInStream failed: %s\n", hipGetErrorString(cuda_status));
        return 1;
    }

    return 0;
}

int StreamTest::TestCreateStreamWithPriority() {
    hipError_t cuda_status = CreateStreamWithPriority();
    if (cuda_status != hipSuccess) {
        fprintf(stderr, "CreateStreamWithPriority failed: %s\n", hipGetErrorString(cuda_status));
        return 1;
    }

    return 0;
}