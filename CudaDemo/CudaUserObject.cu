#include "CudaUserObject.cuh"

//// Library API with pool allocation
//void libraryWork(hipStream_t stream) {
//    auto& resource = pool.claimTemporaryResource();
//    resource.waitOnReadyEventInStream(stream);
//    launchWork(stream, resource);
//    resource.recordReadyEvent(stream);
//}
//// Library API with asynchronous resource deletion
//void libraryWork(hipStream_t stream) {
//    Resource* resource = new Resource(...);
//    launchWork(stream, resource);
//    hipStreamAddCallback(
//        stream,
//        [](hipStream_t, hipError_t, void* resource) {
//            delete static_cast<Resource*>(resource);
//        },
//        resource,
//            0);
//    // Error handling considerations not shown
//}

int CudaUserObject::ManageGraph() {
    //hipGraph_t graph;  // Preexisting graph

    //Object* object = new Object;  // C++ object with possibly nontrivial destructor
    //hipUserObject_t cuObject;
    //hipUserObjectCreate(
    //    &cuObject,
    //    object,  // Here we use a CUDA-provided template wrapper for this API,
    //             // which supplies a callback to delete the C++ object pointer
    //    1,  // Initial refcount
    //    hipUserObjectNoDestructorSync  // Acknowledge that the callback cannot be
    //                                    // waited on via CUDA
    //);
    //hipGraphRetainUserObject(
    //    graph,
    //    cuObject,
    //    1,  // Number of references
    //    hipGraphUserObjectMove  // Transfer a reference owned by the caller (do
    //                             // not modify the total reference count)
    //);
    //// No more references owned by this thread; no need to call release API
    //hipGraphExec_t graphExec;
    //hipGraphInstantiate(&graphExec, graph, nullptr, nullptr, 0);  // Will retain a
    //                                                               // new reference
    //hipGraphDestroy(graph);  // graphExec still owns a reference
    //hipGraphLaunch(graphExec, 0);  // Async launch has access to the user objects
    //hipGraphExecDestroy(graphExec);  // Launch is not synchronized; the release
    //                                  // will be deferred if needed
    //hipStreamSynchronize(0);  // After the launch is synchronized, the remaining
    //                           // reference is released and the destructor will
    //                           // execute. Note this happens asynchronously.
    //// If the destructor callback had signaled a synchronization object, it would
    //// be safe to wait on it at this point.
    return 0;
}