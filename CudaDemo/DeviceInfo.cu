#include "hip/hip_runtime.h"
#include "DeviceInfo.cuh"

#include <stdio.h>

__global__ void DeviceDemoKernel(float* inputDevPtr, int data_size) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < data_size)
        inputDevPtr[i] = inputDevPtr[i] + 1;
}

hipError_t DeviceInfo::GetDeviceList() {
    hipError_t cuda_status;
    int deviceCount;
    cuda_status = hipGetDeviceCount(&deviceCount);
    if (cuda_status != hipSuccess) {
        fprintf(stderr, "hipGetDeviceCount failed: %s\n", hipGetErrorString(cuda_status));
        goto Error;
    }
    int device;
    for (device = 0; device < deviceCount; ++device) {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, device);
        printf("Device %d has compute capability %d.%d.\n",
            device, deviceProp.major, deviceProp.minor);
    }
    cuda_status = hipGetLastError();
    if (cuda_status != hipSuccess) {
        fprintf(stderr, "Enumerate device failed: %s\n", hipGetErrorString(cuda_status));
        goto Error;
    }
Error:
    return cuda_status;
}

hipError_t DeviceInfo::SetDevice() {
    hipError_t cuda_status;

    size_t size = 1024 * sizeof(float);
    
    cuda_status = hipSetDevice(0);            // Set device 0 as current
    float* p0;
    cuda_status = hipMalloc(&p0, size);       // Allocate memory on device 0
    DeviceDemoKernel << <1000, 128 >> > (p0, 1024); // Launch kernel on device 0
    
    cuda_status = hipSetDevice(1);            // Set device 1 as current
    float* p1;
    cuda_status = hipMalloc(&p1, size);       // Allocate memory on device 1
    DeviceDemoKernel << <1000, 128 >> > (p1, 1024); // Launch kernel on device 1

    cuda_status = hipGetLastError();
    if (cuda_status != hipSuccess) {
        fprintf(stderr, "Set different device failed: %s\n", hipGetErrorString(cuda_status));
        goto Error;
    }
Error:
    return cuda_status;
}

hipError_t DeviceInfo::SetStreamOnMultiDevice() {
    hipError_t cuda_status;

    size_t size = 1024 * sizeof(float);

    cuda_status = hipSetDevice(0);               // Set device 0 as current
    float* p0;
    cuda_status = hipMalloc(&p0, size);       // Allocate memory on device 0
    hipStream_t s0;
    cuda_status = hipStreamCreate(&s0);          // Create stream s0 on device 0
    DeviceDemoKernel << <1000, 128, 0, s0 >> > (p0, 1024); // Launch kernel on device 0 in s0

    cuda_status = hipSetDevice(1);               // Set device 1 as current
    float* p1;
    cuda_status = hipMalloc(&p1, size);       // Allocate memory on device 1
    hipStream_t s1;
    cuda_status = hipStreamCreate(&s1);          // Create stream s1 on device 1
    DeviceDemoKernel << <1000, 128, 0, s1 >> > (p1, 1024); // Launch kernel on device 1 in s1

    // This kernel launch will fail:
    DeviceDemoKernel << <1000, 128, 0, s0 >> > (p1, 1024); // Launch kernel on device 1 in s0

    cuda_status = hipGetLastError();
    if (cuda_status != hipSuccess) {
        fprintf(stderr, "Set stream on multi-device failed: %s\n", hipGetErrorString(cuda_status));
        goto Error;
    }
Error:
    return cuda_status;
}

hipError_t DeviceInfo::P2PMemoryAccess() {
    hipError_t cuda_status;
    
    hipSetDevice(0);                   // Set device 0 as current
    float* p0;
    size_t size = 1024 * sizeof(float);
    hipMalloc(&p0, size);              // Allocate memory on device 0
    DeviceDemoKernel << <1000, 128 >> > (p0, 1024);        // Launch kernel on device 0
    hipSetDevice(1);                   // Set device 1 as current
    hipDeviceEnablePeerAccess(0, 0);   // Enable peer-to-peer access
                                       // with device 0

    // Launch kernel on device 1
    // This kernel launch can access memory on device 0 at address p0
    DeviceDemoKernel << <1000, 128 >> > (p0, 1024);

    hipDeviceSynchronize();

    cuda_status = hipGetLastError();
    if (cuda_status != hipSuccess) {
        fprintf(stderr, "P2PMemoryAccess on multi-device failed: %s\n", hipGetErrorString(cuda_status));
        goto Error;
    }
Error:
    return cuda_status;
}

int DeviceInfo::TestGetDeviceList() {
    hipError_t cuda_status = GetDeviceList();
    if (cuda_status != hipSuccess) {
        fprintf(stderr, "GetDeviceList failed: %s\n", hipGetErrorString(cuda_status));
        return 1;
    }

    return 0;
}

int DeviceInfo::TestSetDevice() {
    hipError_t cuda_status = SetDevice();
    if (cuda_status != hipSuccess) {
        fprintf(stderr, "SetDevice failed: %s\n", hipGetErrorString(cuda_status));
        return 1;
    }

    return 0;
}

int DeviceInfo::TestSetStreamOnMultiDevice() {
    hipError_t cuda_status = SetStreamOnMultiDevice();
    if (cuda_status != hipSuccess) {
        fprintf(stderr, "SetStreamOnMultiDevice failed: %s\n", hipGetErrorString(cuda_status));
        return 1;
    }

    return 0;
}