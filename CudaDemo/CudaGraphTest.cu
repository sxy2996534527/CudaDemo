#include "hip/hip_runtime.h"
#include "CudaGraphTest.cuh"

#include <stdio.h>

__global__ void AddKernel1(int* c, const int* a, const int* b)
{
	int i = threadIdx.x;
	c[i] = a[i] + b[i];
}

__global__ void AddKernel2(int* c, const int* a, const int* b)
{
	int i = threadIdx.x;
	c[i] = a[i] + b[i];
}

__global__ void AddKernel3(int* c, const int* a, const int* b)
{
	int i = threadIdx.x;
	c[i] = a[i] + b[i];
}

__global__ void AddKernel4(int* c, const int* a, const int* b)
{
	int i = threadIdx.x;
	c[i] = a[i] + b[i];
}

hipError_t CudaGraphTest::CreateGraphWithAPI() {
	hipError_t cuda_status;

	// Create the graph - it starts out empty
	hipGraph_t graph;
	hipGraphCreate(&graph, 0);

	// For the purpose of this example, we'll create
	// the nodes separately from the dependencies to
	// demonstrate that it can be done in two stages.
	// Note that dependencies can also be specified 
	// at node creation. 
	hipGraphNode_t a, b, c, d;
	hipKernelNodeParams nodeParams;
	hipGraphAddKernelNode(&a, graph, NULL, 0, &nodeParams);
	hipGraphAddKernelNode(&b, graph, NULL, 0, &nodeParams);
	hipGraphAddKernelNode(&c, graph, NULL, 0, &nodeParams);
	hipGraphAddKernelNode(&d, graph, NULL, 0, &nodeParams);

	// Now set up dependencies on each node
	hipGraphAddDependencies(graph, &a, &b, 1);     // A->B
	hipGraphAddDependencies(graph, &a, &c, 1);     // A->C
	hipGraphAddDependencies(graph, &b, &d, 1);     // B->D
	hipGraphAddDependencies(graph, &c, &d, 1);     // C->D

	cuda_status = hipGetLastError();
	if (cuda_status != hipSuccess) {
		fprintf(stderr, "CreateGraphWithAPI failed: %s\n", hipGetErrorString(cuda_status));
		goto Error;
	}
Error:
	return cuda_status;
}

hipError_t CudaGraphTest::CreateGraphWithStreamCapture() {
	hipError_t cuda_status;

	//test data
	int size = 512;
	int data_size = 512 * sizeof(int);
	int* a = new int[size]();
	int* b = new int[size]();
	int* c = new int[size]();

	for (int i = 0; i < size; i++) {
		a[i] = i;
		b[i] = 2 * i;
	}

	int* d_a = 0;
	int* d_b = 0;
	int* d_c = 0;
	hipMalloc(&d_a, data_size);
	hipMalloc(&d_b, data_size);
	hipMalloc(&d_c, data_size);

	hipMemcpy(d_a, a, data_size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, data_size, hipMemcpyHostToDevice);

	hipStream_t stream;
	cuda_status = hipStreamCreate(&stream);

	hipGraph_t graph;

	hipStreamBeginCapture(stream, hipStreamCaptureModeGlobal);

	AddKernel1 << < 1, size, 0, stream >> > (d_c, d_a, d_b);
	AddKernel2 << < 1, size, 0, stream >> > (d_c, d_a, d_b);
	//libraryCall(stream);
	AddKernel3 << < 1, size, 0, stream >> > (d_c, d_a, d_b);

	hipStreamEndCapture(stream, &graph);
	
	cuda_status = hipGetLastError();
	if (cuda_status != hipSuccess) {
		fprintf(stderr, "CreateGraphWithAPI failed: %s\n", hipGetErrorString(cuda_status));
		goto Error;
	}

	// Instantiate and launch the graph
	hipGraphExec_t graphExec;
	hipGraphInstantiate(&graphExec, graph, NULL, NULL, 0);
	hipGraphLaunch(graphExec, stream);

	// Wait for completion
	hipStreamSynchronize(stream);

	// Copy output vector from GPU buffer to host memory.
	cuda_status = hipMemcpy(c, d_c, data_size, hipMemcpyDeviceToHost);
	if (cuda_status != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}


Error:
	// Clean up
	hipGraphExecDestroy(graphExec);
	hipGraphDestroy(graph);
	hipStreamDestroy(stream);
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);

	delete[] a;
	delete[] b;
	delete[] c;

	return cuda_status;
}

hipError_t CudaGraphTest::CrossStreamDependencyAndEvent() {
	hipError_t cuda_status;

	//test data
	int size = 512;
	int data_size = 512 * sizeof(int);
	int* a = new int[size]();
	int* b = new int[size]();
	int* c = new int[size]();

	for (int i = 0; i < size; i++) {
		a[i] = i;
		b[i] = 2 * i;
	}

	int* d_a = 0;
	int* d_b = 0;
	int* d_c = 0;
	hipMalloc(&d_a, data_size);
	hipMalloc(&d_b, data_size);
	hipMalloc(&d_c, data_size);

	hipMemcpy(d_a, a, data_size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, data_size, hipMemcpyHostToDevice);

	hipStream_t stream1, stream2;
	cuda_status = hipStreamCreate(&stream1);
	cuda_status = hipStreamCreate(&stream2);

	// stream1 is the origin stream
	hipStreamBeginCapture(stream1, hipStreamCaptureModeGlobal);

	AddKernel1 << < 1, size, 0, stream1 >> > (d_c, d_a, d_b);

	// Fork into stream2
	hipEvent_t event1;
	hipEventCreate(&event1);
	hipEventRecord(event1, stream1);
	hipStreamWaitEvent(stream2, event1);

	AddKernel2 << < 1, size, 0, stream1 >> > (d_c, d_a, d_b);
	AddKernel3 << < 1, size, 0, stream2 >> > (d_c, d_a, d_b);

	// Join stream2 back to origin stream (stream1)
	hipEvent_t event2;
	hipEventCreate(&event2);
	hipEventRecord(event2, stream2);
	hipStreamWaitEvent(stream1, event2);

	AddKernel4 << < 1, size, 0, stream1 >> > (d_c, d_a, d_b);

	// End capture in the origin stream
	hipGraph_t graph;
	hipStreamEndCapture(stream1, &graph);

	// stream1 and stream2 no longer in capture mode  

	cuda_status = hipGetLastError();
	if (cuda_status != hipSuccess) {
		fprintf(stderr, "CrossStreamDependencyAndEvent failed: %s\n", hipGetErrorString(cuda_status));
		goto Error;
	}

	// Instantiate and launch the graph
	hipGraphExec_t graphExec;
	hipGraphInstantiate(&graphExec, graph, NULL, NULL, 0);
	hipGraphLaunch(graphExec, stream1);

	// Wait for completion
	hipStreamSynchronize(stream1);
	hipStreamSynchronize(stream2);

	// Copy output vector from GPU buffer to host memory.
	cuda_status = hipMemcpy(c, d_c, data_size, hipMemcpyDeviceToHost);
	if (cuda_status != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}


Error:
	// Clean up
	hipGraphExecDestroy(graphExec);
	hipGraphDestroy(graph);
	hipStreamDestroy(stream1);
	hipStreamDestroy(stream2);
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);

	delete[] a;
	delete[] b;
	delete[] c;

	return cuda_status;
}

hipError_t CudaGraphTest::UpdateGlobalGraph() {
	hipError_t cuda_status;

	hipGraphExec_t graphExec = NULL;

	hipStream_t stream;
	hipStreamCreate(&stream);

	//test data
	int size = 512;
	int data_size = 512 * sizeof(int);
	int* a = new int[size]();
	int* b = new int[size]();
	int* c = new int[size]();

	for (int i = 0; i < size; i++) {
		a[i] = i;
		b[i] = 2 * i;
	}

	int* d_a = 0;
	int* d_b = 0;
	int* d_c = 0;
	hipMalloc(&d_a, data_size);
	hipMalloc(&d_b, data_size);
	hipMalloc(&d_c, data_size);

	hipMemcpy(d_a, a, data_size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, data_size, hipMemcpyHostToDevice);

	for (int i = 0; i < 10; i++) {
		hipGraph_t graph;
		hipGraphExecUpdateResult updateResult;
		hipGraphNode_t errorNode;

		// In this example we use stream capture to create the graph.
		// You can also use the Graph API to produce a graph.
		hipStreamBeginCapture(stream, hipStreamCaptureModeGlobal);

		// Call a user-defined, stream based workload, for example
		AddKernel1 << < 1, size, 0, stream >> > (d_c, d_a, d_b);
		AddKernel2 << < 1, size, 0, stream >> > (d_c, d_a, d_b);
		//libraryCall(stream);
		AddKernel3 << < 1, size, 0, stream >> > (d_c, d_a, d_b);

		hipStreamEndCapture(stream, &graph);

		// If we've already instantiated the graph, try to update it directly
		// and avoid the instantiation overhead
		if (graphExec != NULL) {
			// If the graph fails to update, errorNode will be set to the
			// node causing the failure and updateResult will be set to a
			// reason code.
			hipGraphExecUpdate(graphExec, graph, &errorNode, &updateResult);
		}

		// Instantiate during the first iteration or whenever the update
		// fails for any reason
		if (graphExec == NULL || updateResult != hipGraphExecUpdateSuccess) {

			// If a previous update failed, destroy the hipGraphExec_t
			// before re-instantiating it
			if (graphExec != NULL) {
				hipGraphExecDestroy(graphExec);
			}
			// Instantiate graphExec from graph. The error node and
			// error message parameters are unused here.
			hipGraphInstantiate(&graphExec, graph, NULL, NULL, 0);
		}

		hipGraphDestroy(graph);
		hipGraphLaunch(graphExec, stream);
		hipStreamSynchronize(stream);
	}
	cuda_status = hipGetLastError();
	if (cuda_status != hipSuccess) {
		fprintf(stderr, "UpdateGlobalGraph failed: %s\n", hipGetErrorString(cuda_status));
		goto Error;
	}
	// Copy output vector from GPU buffer to host memory.
	cuda_status = hipMemcpy(c, d_c, data_size, hipMemcpyDeviceToHost);
	if (cuda_status != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}
Error:
	// Clean up
	hipGraphExecDestroy(graphExec);
	//hipGraphDestroy(graph);
	hipStreamDestroy(stream);
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);

	delete[] a;
	delete[] b;
	delete[] c;
	return cuda_status;
}

int CudaGraphTest::TestCreateGraphWithStreamCapture() {
	hipError_t cuda_status = CreateGraphWithStreamCapture();
	if (cuda_status != hipSuccess) {
		fprintf(stderr, "CreateGraphWithStreamCapture failed: %s\n", hipGetErrorString(cuda_status));
		return 1;
	}

	return 0;
}

int CudaGraphTest::TestCrossStreamDependencyAndEvent() {
	hipError_t cuda_status = CrossStreamDependencyAndEvent();
	if (cuda_status != hipSuccess) {
		fprintf(stderr, "CrossStreamDependencyAndEvent failed: %s\n", hipGetErrorString(cuda_status));
		return 1;
	}

	return 0;
}


int CudaGraphTest::TestUpdateGlobalGraph() {
	hipError_t cuda_status = UpdateGlobalGraph();
	if (cuda_status != hipSuccess) {
		fprintf(stderr, "UpdateGlobalGraph failed: %s\n", hipGetErrorString(cuda_status));
		return 1;
	}

	return 0;
}